#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <math.h>
#include <algorithm>
#include <iostream>
#include <time.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "helper/inc/helper_functions.h" // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include "helper/inc/hip/hip_runtime_api.h" // helper functions for CUDA error check

// Convolution Mask Dimension
#define MASK_DIM 3
#define OFFSET (MASK_DIM/2)

#define TILE_WIDTH 16
#define RADIUS 2
#define BLOCK_WIDTH (TILE_WIDTH+(2*RADIUS))


#define DIAMETER (RADIUS*2+1) // filter diameter
#define SIZE (RADIUS*DIAMETER) // filter size


__constant__ float mask[MASK_DIM * MASK_DIM];

// allocate mask in constant memory
__constant__ float d_mask_global[MASK_DIM * MASK_DIM];
__constant__ float d_mask_shared[MASK_DIM][MASK_DIM];

// print 1D array function
void printArray(float *array, int width) {
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            printf("%3.6f ", array[(i * width) + j]);
        }
        printf("\n");
    }
}

// 2D convolution using global and constant memory
__global__ void global_convolution(float *d_Data, float *d_result, int width, int height) {
  // calculate the row and column index to compute for each thread
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Starting index for convolution so we can ignore the padded area
  int i_row = row - OFFSET;
  int i_col = col - OFFSET;

  // convolution value to be calculated for each pixel's row and column
  double value = 0;
  // iterate over all rows and column using the mask dimension.
  // this will calulate all the neighbours and origin pixel and sum these values to give
  // us the value of the origin pixel
  for (int i = 0; i < MASK_DIM; i++) {
    for (int j = 0; j < MASK_DIM; j++) {
      if ((i_row + i) >= 0 && (i_row + i) < height && (i_col + j) >= 0 && (i_col + j) < width) {
        //   printf("martix %d x %d value: %3.6 --- Mask value: %3.6f \n",i,j, matrix[(start_row + i) * N + (start_col + j)], d_mask[i * MASK_DIM + j]);
        value += d_Data[(i_row + i) * width + (i_col + j)] * d_mask_global[i * MASK_DIM + j];
      }
    }
  }
  // write back convolution result
  d_result[row * width + col] = value;
}

// 2D convolution using shared and constant memory
__global__ void shared(float *d_data, float *d_result, unsigned int width, unsigned int height) {
  
  // create tile in shared memrory for the convolution
  __shared__ float shared[TILE_WIDTH + MASK_DIM -1][TILE_WIDTH + MASK_DIM -1];
  
  // for simplicity to use threadIdx
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // get row and column index of pixels in the tile
  int row = by * TILE_WIDTH + ty;
  int col = bx * TILE_WIDTH + tx;
  
  // row and column index to stat from so we can ignore the padded area.
  int row_i = row - OFFSET;
  int col_i = col - OFFSET;

  // __syncthreads();
  // load the tile pixels from the global memory into shared memory
  // this will help us to reduce global memory access by the factor of 1/TILE_WIDTH
  // ignore any pixels which are out-of-bounds (i.e. padded area)
  if ((row_i>=0) && (row_i < height) && (col_i >=0) && (col_i < width)){
    shared[ty][tx] = d_data[row_i*width+col_i];
  } else {
    shared[ty][tx]=0;
  }

  // thread barrier to wait for all the threads to finish loading from
  // global memory to shared memory
  __syncthreads();

  float output =0;
  // only certain threads calculate the result
  // Elementwise multiplication of pixel and mask values and add all of the neighbours
  // to get output of one pixel (origin pixel)
  if (ty < TILE_WIDTH &&  tx < TILE_WIDTH){
    for (int i=0; i< MASK_DIM; i++){
      for (int j=0; j<MASK_DIM; j++){
        output += d_mask_shared[i][j] * shared[i+ty][j+tx];
      }
    }
  }

  // thread barrier to wait for all threads to finish convolution
  __syncthreads();

  // write output to the results image
  if (row < height && col < width){
    d_result[row * width + col] = output;
  }
}

__global__ void shared_convolution(float* dData, float* dResult, unsigned int width, unsigned int height){

  // create tile in shared memrory for the convolution
  __shared__ float shared[BLOCK_WIDTH * BLOCK_WIDTH];

    // for simplicity to use threadIdx
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // get row and column index of pixels in the tile
    int col = bx * TILE_WIDTH + tx - RADIUS;
    int row = by * TILE_WIDTH + ty - RADIUS;

    // Find the last and first pixel locations within the image
    col = max(0, col);
    col = min(col, width-1);
    row = max(row, 0);
    row = min(row, height-1);

    // load the tile pixels from the global memory into shared memory
    // this will help us to reduce global memory access by the factor of 1/TILE_WIDTH
    // ignore any pixels which are out-of-bounds (i.e. padded area)
    unsigned int index = row * width + col;
    unsigned int block_index = ty * blockDim.y + tx;
    shared[block_index] = dData[index];

    // thread barrier to wait for all the threads to finish loading from
    // global memory to shared memory
    __syncthreads();
  
    // Elementwise multiplication of pixel and mask values and add all of the values within the mask
    // range to get output value of one pixel. Verify that we are not working out-of-bounds of the image
    // We will iterate over rows and columns within the mask dimensions (i.e. all the neighbours)
    float value = 0;
    if (((tx >= RADIUS) && (tx < BLOCK_WIDTH-RADIUS)) && ((ty>=RADIUS) && (ty<=BLOCK_WIDTH-RADIUS))){
      for(int i = 0; i<MASK_DIM; i++){
          for(int j = 0; j<MASK_DIM; j++){ 
            value += shared[block_index+(i*blockDim.x)+j] * mask[i*3+j];
          }
      }
      dResult[index] = value;
  }
}

int main(int argc, char **argv){

  // image file names as input
  // const char *imageFilename = "image21.pgm";
  const char *imageFilename = "lena_bw.pgm";
  // const char *imageFilename = "man.pgm";
  // const char *imageFilename = "mandrill.pgm";

  // load image from disk
  float *hData = NULL;
  unsigned int width, height;
  char *imagePath = sdkFindFilePath(imageFilename, argv[0]);

  if (imagePath == NULL)
  {
      printf("Unable to source image file: %s\n", imageFilename);
      exit(EXIT_FAILURE);
  }

  sdkLoadPGM(imagePath, &hData, &width, &height);

  unsigned int image_size = width * height * sizeof(float);
  printf("Loaded '%s', %d x %d pixels\n", imageFilename, width, height);

  // printf("Input image \n");
  // printArray(hData, 10);

  // allocate memory for mask
  size_t mask_size = MASK_DIM * MASK_DIM * sizeof(float);

  // Allocate memory for h_result image
  float *h_result = (float *)malloc(sizeof(float) * width * height);;

  //-------------- Initialise Masks --------------//
  // edge detection
  // float h_mask[MASK_DIM][MASK_DIM] = {
  //   {-1, 0, 1},
  //   {-2, 0, 2},
  //   {-1, 0, 1},
  // };

  // shapenning filter
  // float h_mask[MASK_DIM][MASK_DIM] = {
  //   {-1, -1, -1},
  //   {-1,  9, -1},
  //   {-1, -1, -1},
  // };
  float constant_mem_mask[MASK_DIM * MASK_DIM]= {-1, -1, -1, -1, 9, -1, -1, -1, -1};

  // averaging filter
  // float h_mask[MASK_DIM][MASK_DIM] = {
  //   {1, 1, 1},
  //   {1, 1, 1},
  //   {1, 1, 1},
  // };

  //-------------- CUDA --------------//
  // Allocate device memory
  float *d_image;
  float *d_result;
  checkCudaErrors(hipMalloc((void**)&d_image, sizeof(float) * image_size));
  checkCudaErrors(hipMalloc((void**)&d_result, sizeof(float) * image_size));

  // Copy data to the device
  checkCudaErrors(hipMemcpy(d_image, hData, image_size, hipMemcpyHostToDevice));
  // checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_mask_shared), h_mask, mask_size));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(mask), constant_mem_mask, mask_size));


  // CUDA timing of event
  hipEvent_t shared_start, shared_stop;
  hipEventCreate(&shared_start);
  hipEventCreate(&shared_stop);

  // Calculate grid dimensions for dimGrid
  int BLOCKS = (width-1)/TILE_WIDTH+1;
  // int BLOCKS = (width+TILE_WIDTH-1)/TILE_WIDTH;
  // Dimension for the kernel launch
  // dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
  dim3 dimGrid(BLOCKS, BLOCKS);
    // dim3 dimGrid(32, 32);
    dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);

  //-------------- Shared Convolution --------------//
  // start the shared memory kernel
    hipEventRecord(shared_start);
    shared_convolution<<<dimGrid, dimBlock>>>(d_image, d_result, width, height);
    hipEventRecord(shared_stop); 
    hipEventSynchronize(shared_stop);
        
    float shared_elapsedTime = 0;
    hipEventElapsedTime(&shared_elapsedTime, shared_start, shared_stop);

    hipEventDestroy(shared_start);
    hipEventDestroy(shared_stop);

    // Copy the h_result back to the CPU
    checkCudaErrors(hipMemcpy(h_result, d_result, image_size, hipMemcpyDeviceToHost));
    //   printf("Result image \n");
    //   printArray(h_result, 10);

  //-------------- Write Convolution Results to output image --------------//
  char outputFilename[1024];
  strcpy(outputFilename, imagePath);
  strcpy(outputFilename + strlen(imagePath) - 4, "_shared.pgm");
  sdkSavePGM(outputFilename, h_result, width, height);
  printf("Wrote '%s'\n", outputFilename);

  //-------------- CUDA Performance Metrics --------------//
  float num_ops= width * height; // every element swap once

  float shared_throughput = num_ops / (shared_elapsedTime / 1000.0f) / 1000000000.0f;
  
  std::cout << "Matrix size: " << width << "x" << height << std::endl;
  std::cout << "Tile size: " << TILE_WIDTH << "x" << TILE_WIDTH << std::endl;

  printf("Shared Memory Time elpased: %3.6f ms \n", shared_elapsedTime);

  std::cout << "Throughput of shared memory kernel: " << shared_throughput << " GFLOPS" << std::endl;


  //-------------- CUDA Free Memory --------------//
  // Free the memory we allocated
  free(imagePath);
  free(h_result);

  //   checkCudaErrors(hipFree(h_mask));
  //   checkCudaErrors(hipFree(d_mask));
  checkCudaErrors(hipFree(d_image));
  checkCudaErrors(hipFree(d_result));

  return 0;
}
