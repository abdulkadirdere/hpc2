#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <math.h>
#include <algorithm>
#include <iostream>
#include <time.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "helper/inc/helper_functions.h" // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include "helper/inc/hip/hip_runtime_api.h" // helper functions for CUDA error check

// Convolution Mask Dimension
#define MASK_DIM 3
#define OFFSET (MASK_DIM/2)

#define TILE_WIDTH 12
#define BLOCK_WIDTH (TILE_WIDTH + MASK_DIM -1)


// allocate mask in constant memory
__constant__ float d_mask_global[MASK_DIM * MASK_DIM];
__constant__ float d_mask_shared[MASK_DIM][MASK_DIM];

// print 1D array function
void printArray(float *array, int width) {
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            printf("%3.6f ", array[(i * width) + j]);
        }
        printf("\n");
    }
}

// 2D convolution using global and constant memory
__global__ void global_convolution(float *d_Data, float *d_result, int width, int height) {
  // calculate the row and column index to compute for each thread
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Starting index for convolution so we can ignore the padded area
  int i_row = row - OFFSET;
  int i_col = col - OFFSET;

  // convolution value to be calculated for each pixel's row and column
  double value = 0;
  // iterate over all rows and column using the mask dimension.
  // this will calulate all the neighbours and origin pixel and sum these values to give
  // us the value of the origin pixel
  for (int i = 0; i < MASK_DIM; i++) {
    for (int j = 0; j < MASK_DIM; j++) {
      if ((i_row + i) >= 0 && (i_row + i) < height && (i_col + j) >= 0 && (i_col + j) < width) {
        //   printf("martix %d x %d value: %3.6 --- Mask value: %3.6f \n",i,j, matrix[(start_row + i) * N + (start_col + j)], d_mask[i * MASK_DIM + j]);
        value += d_Data[(i_row + i) * width + (i_col + j)] * d_mask_global[i * MASK_DIM + j];
      }
    }
  }
  // write back convolution result
  d_result[row * width + col] = value;
}

// 2D convolution using shared and constant memory
__global__ void shared_conv(float *d_data, float *d_result, unsigned int width, unsigned int height) {
  
  // create tile in shared memrory for the convolution
  __shared__ float shared[TILE_WIDTH + MASK_DIM -1][TILE_WIDTH + MASK_DIM -1];
  
  // for simplicity to use threadIdx
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // get row and column index of pixels in the tile
  int row = by * TILE_WIDTH + ty;
  int col = bx * TILE_WIDTH + tx;
  
  // row and column index to stat from so we can ignore the padded area.
  int row_i = row - OFFSET;
  int col_i = col - OFFSET;

  // __syncthreads();
  // load the tile pixels from the global memory into shared memory
  // this will help us to reduce global memory access by the factor of 1/TILE_WIDTH
  // ignore any pixels which are out-of-bounds (i.e. padded area)
  if ((row_i>=0) && (row_i < height) && (col_i >=0) && (col_i < width)){
    shared[ty][tx] = d_data[row_i*width+col_i];
  } else {
    shared[ty][tx]=0;
  }

  // thread barrier to wait for all the threads to finish loading from
  // global memory to shared memory
  __syncthreads();

  float output =0;
  // only certain threads calculate the result
  // Elementwise multiplication of pixel and mask values and add all of the neighbours
  // to get output of one pixel (origin pixel)
  if (ty < TILE_WIDTH &&  tx < TILE_WIDTH){
    for (int i=0; i< MASK_DIM; i++){
      for (int j=0; j<MASK_DIM; j++){
        output += d_mask_shared[i][j] * shared[i+ty][j+tx];
      }
    }
  }

  // thread barrier to wait for all threads to finish convolution
  __syncthreads();

  // write output to the results image
  if (row < height && col < width){
    d_result[row * width + col] = output;
  }
}



int main(int argc, char **argv){

  // image file names as input
  // const char *imageFilename = "image21.pgm";
  const char *imageFilename = "lena_bw.pgm";
  // const char *imageFilename = "man.pgm";
  // const char *imageFilename = "mandrill.pgm";

  // load image from disk
  float *hData = NULL;
  unsigned int width, height;
  char *imagePath = sdkFindFilePath(imageFilename, argv[0]);

  if (imagePath == NULL)
  {
      printf("Unable to source image file: %s\n", imageFilename);
      exit(EXIT_FAILURE);
  }

  sdkLoadPGM(imagePath, &hData, &width, &height);

  unsigned int image_size = width * height * sizeof(float);
  printf("Loaded '%s', %d x %d pixels\n", imageFilename, width, height);

  // printf("Input image \n");
  // printArray(hData, 10);

  // allocate memory for mask
  size_t mask_size = MASK_DIM * MASK_DIM * sizeof(float);

  // Allocate memory for h_result image
  float *h_result = (float *)malloc(sizeof(float) * width * height);;

  //-------------- Initialise Masks --------------//
  // edge detection
  float h_mask[MASK_DIM][MASK_DIM] = {
    {-1, 0, 1},
    {-2, 0, 2},
    {-1, 0, 1},
  };

  // shapenning filter
  // float h_mask[MASK_DIM][MASK_DIM] = {
  //   {-1, -1, -1},
  //   {-1,  9, -1},
  //   {-1, -1, -1},
  // };

  // averaging filter
  // float h_mask[MASK_DIM][MASK_DIM] = {
  //   {1, 1, 1},
  //   {1, 1, 1},
  //   {1, 1, 1},
  // };

  //-------------- CUDA --------------//
  // Allocate device memory
  float *d_image;
  float *d_result;
  checkCudaErrors(hipMalloc((void**)&d_image, sizeof(float) * image_size));
  checkCudaErrors(hipMalloc((void**)&d_result, sizeof(float) * image_size));

  // Copy data to the device
  checkCudaErrors(hipMemcpy(d_image, hData, image_size, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_mask_shared), h_mask, mask_size));

  // CUDA timing of event
  hipEvent_t shared_start, shared_stop;
  hipEventCreate(&shared_start);
  hipEventCreate(&shared_stop);

  // Calculate grid dimensions for dimGrid
  int BLOCKS = (width-1)/TILE_WIDTH+1;
  // int BLOCKS = (width+TILE_WIDTH-1)/TILE_WIDTH;
  // Dimension for the kernel launch
  dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
  dim3 dimGrid(BLOCKS, BLOCKS);

  //-------------- Shared Convolution --------------//
  // start the shared memory kernel
    hipEventRecord(shared_start);
    shared_conv<<<dimGrid, dimBlock>>>(d_image, d_result, width, height);
    hipEventRecord(shared_stop); 
    hipEventSynchronize(shared_stop);
        
    float shared_elapsedTime = 0;
    hipEventElapsedTime(&shared_elapsedTime, shared_start, shared_stop);

    hipEventDestroy(shared_start);
    hipEventDestroy(shared_stop);

    // Copy the h_result back to the CPU
    checkCudaErrors(hipMemcpy(h_result, d_result, image_size, hipMemcpyDeviceToHost));
    //   printf("Result image \n");
    //   printArray(h_result, 10);

  //-------------- Write Convolution Results to output image --------------//
  char outputFilename[1024];
  strcpy(outputFilename, imagePath);
  strcpy(outputFilename + strlen(imagePath) - 4, "_shared.pgm");
  sdkSavePGM(outputFilename, h_result, width, height);
  printf("Wrote '%s'\n", outputFilename);

  //-------------- CUDA Performance Metrics --------------//
  float num_ops= width * height; // every element swap once

  float shared_throughput = num_ops / (shared_elapsedTime / 1000.0f) / 1000000000.0f;
  
  std::cout << "Matrix size: " << width << "x" << height << std::endl;
  std::cout << "Tile size: " << TILE_WIDTH << "x" << TILE_WIDTH << std::endl;

  printf("Shared Memory Time elpased: %3.6f ms \n", shared_elapsedTime);

  std::cout << "Throughput of shared memory kernel: " << shared_throughput << " GFLOPS" << std::endl;





  //-------------- CUDA Free Memory --------------//
  // Free the memory we allocated
  free(imagePath);
  free(h_result);

  //   checkCudaErrors(hipFree(h_mask));
  //   checkCudaErrors(hipFree(d_mask));
  checkCudaErrors(hipFree(d_image));
  checkCudaErrors(hipFree(d_result));

  return 0;
}
