#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdlib>

#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <math.h>
#include <algorithm>
#include <iostream>
#include <time.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "helper/inc/helper_functions.h" // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include "helper/inc/hip/hip_runtime_api.h" // helper functions for CUDA error check

// Convolution Mask Dimension
#define MASK_DIM 3
#define OFFSET (MASK_DIM/2)

#define TILE_WIDTH 12
// TILE_WIDTH + MASK_DIM -1
#define BLOCK_WIDTH (TILE_WIDTH + MASK_DIM -1)



// allocate mask in constant memory
__constant__ float d_mask[MASK_DIM * MASK_DIM];
__constant__ float d_M[MASK_DIM][MASK_DIM];

// print 1D array function
void printArray(float *array, int width) {
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            printf("%3.6f ", array[(i * width) + j]);
        }
        printf("\n");
    }
}


__global__ void global_convolution(float *d_Data, float *d_result, int width, int height) {
  // Calculate the global thread positions
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Starting index for calculation
  int start_row = row - OFFSET;
  int start_col = col - OFFSET;

  // convolution value to be calculated for each pixel's row and column
   double value = 0;
  // Iterate over all the rows
  for (int i = 0; i < MASK_DIM; i++) {
    // Go over each column
    for (int j = 0; j < MASK_DIM; j++) {
      // Range check for rowsint
      if ((start_row + i) >= 0 && (start_row + i) < height) {
        // Range check for columns
        if ((start_col + j) >= 0 && (start_col + j) < width) {
        //   printf("martix %d x %d value: %3.6 --- Mask value: %3.6f \n",i,j, matrix[(start_row + i) * N + (start_col + j)], d_mask[i * MASK_DIM + j]);
            value += d_Data[(start_row + i) * width + (start_col + j)] * d_mask[i * MASK_DIM + j];
        }
      }
    }
  }
  // write back convolution result
  d_result[row * width + col] = value;
}


// __global__ void d_filter(float *g_idata, float *g_odata, unsigned int width, unsigned int height) {
//     __shared__ float smem[BLOCK_W*BLOCK_H];

//     int x = blockIdx.x*TILE_W + threadIdx.x;
//     int y = blockIdx.y*TILE_H + threadIdx.y;// clamp to edge of image

//     x = max(0, x);
//     x = min(x, width-1);
//     y = max(y, 0);
//     y = min(y, height-1);
//     int start_row = x - OFFSET;
//     int start_col = y - OFFSET;

//     unsigned int index = y*width + x;
//     unsigned int bindex = threadIdx.y*blockDim.y+threadIdx.x;    // each thread copies its pixel of the block to shared memory
    
//     smem[bindex] = g_idata[index];
//     __syncthreads();

    
//     double value = 0;
//     // only threads inside the apron will write results
//     for (int i = 0; i < MASK_DIM; i++) {
//         // Go over each column
//         for (int j = 0; j < MASK_DIM; j++) {
//           // Range check for rowsint
//           if ((start_row + i) >= 0 && (start_row + i) < height && (start_col + j) >= 0 && (start_col + j) < width) {
//             // Range check for columns
//             // if ((start_col + j) >= 0 && (start_col + j) < width) {
//             //   printf("martix %d x %d value: %3.6 --- Mask value: %3.6f \n",i,j, matrix[(start_row + i) * N + (start_col + j)], d_mask[i * MASK_DIM + j]);
//                 value += smem[(start_row + i) * width + (start_col + j)] * d_mask[i * MASK_DIM + j];
//             // }
//           }
//         }
//       }
//       g_odata[x * width + y] = value;
// }
  


// Initializes an n x n matrix with random numbers
// Takes:
//  m : Pointer to the matrix
//  n : Dimension of the matrix (square)
void init_matrix(float *m, int n) {
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      m[n * i + j] = 1;
    }
  }
}


void verify_result(float *m, float *d_mask, float *result, int N) {
  // Temp value for accumulating results
  float temp;

  // Intermediate value for more readable code
  int offset_r;
  int offset_c;

  // Go over each row
  for (int i = 0; i < N; i++) {
    // Go over each column
    for (int j = 0; j < N; j++) {
      // Reset the temp variable
      temp = 0;

      // Go over each mask row
      for (int k = 0; k < MASK_DIM; k++) {
        // Update offset value for row
        offset_r = i - OFFSET + k;

        // Go over each mask column
        for (int l = 0; l < MASK_DIM; l++) {
          // Update offset value for column
          offset_c = j - OFFSET + l;

          // Range checks if we are hanging off the matrix
          if (offset_r >= 0 && offset_r < N) {
            if (offset_c >= 0 && offset_c < N) {
              // Accumulate partial results
              temp += m[offset_r * N + offset_c] * d_mask[k * MASK_DIM + l];
            }
          }
        }
      }
      // Fail if the results don't match
      assert(result[i * N + j] == temp);
    }
  }
}


__global__ void shared_conv(float *d_data, float *d_result, unsigned int width, unsigned int height) {

  __shared__ float shared[TILE_WIDTH + MASK_DIM -1][TILE_WIDTH + MASK_DIM -1];;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row = blockIdx.y * TILE_WIDTH + ty;
  int col = blockIdx.x * TILE_WIDTH + tx;
  
  int row_i = row - OFFSET;
  int col_i = col - OFFSET;

  // __syncthreads();

  float output =0;
  if ((row_i>=0) && (row_i < height) && (col_i >=0) && (col_i < width)){
    shared[ty][tx] = d_data[row_i*width+col_i];
  } else {
    shared[ty][tx]=0;
  }

  __syncthreads();

  if (ty < TILE_WIDTH &&  tx < TILE_WIDTH){
    for (int i=0; i< MASK_DIM; i++){
      for (int j=0; j<MASK_DIM; j++){
        output += d_M[i][j] * shared[i+ty][j+tx];
      }
    }
  }

  if (row < height && col < width){
    // printf("test output\n");
    d_result[row * width + col] = output;
  }
}



int main(int argc, char **argv){
  // Dimensions of the matrix (2 ^ 10 x 2 ^ 10)
//   int N = 512;
int N = 1<<9;
// printf("%d \n",N);

//   // Size of the matrix (in bytes)
//   size_t image_size = N * N * sizeof(float);


// const char *imageFilename = "image21.pgm";
const char *imageFilename = "lena_bw.pgm";
// const char *imageFilename = "man.pgm";
// const char *imageFilename = "mandrill.pgm";

// load image from disk
float *hData = NULL;
unsigned int width, height;
char *imagePath = sdkFindFilePath(imageFilename, argv[0]);

if (imagePath == NULL)
{
    printf("Unable to source image file: %s\n", imageFilename);
    exit(EXIT_FAILURE);
}

sdkLoadPGM(imagePath, &hData, &width, &height);

unsigned int size = width * height * sizeof(float);
printf("Loaded '%s', %d x %d pixels\n", imageFilename, width, height);

// printf("Input image \n");
// printArray(hData, 10);

// Size of the mask in bytes
// Size of the matrix (in bytes)
size_t image_size = width * height * sizeof(float);
size_t mask_size = MASK_DIM * MASK_DIM * sizeof(float);
// Allocate the matrix and initialize it
float *h_result = (float *)malloc(sizeof(float) * width * height);;

// Allocate the mask and initialize it
float *h_mask;
//   init_matrix(h_mask, MASK_DIM);
// float h_mask[9] = {-1, -2, -1, 0, 0, 0, 1, 2, 1};
//   printArray(h_mask, 7);
// float *h_mask = NULL;
h_mask[MASK_DIM][MASK_DIM] = {
  {-1, 0, 1},
  {-2, 0, 2},
  {-1, 0, 1},
};

//-------------- CUDA --------------//
// Allocate device memory
float *d_image;
float *d_result;
checkCudaErrors(hipMalloc((void**)&d_image, sizeof(float) * image_size));
checkCudaErrors(hipMalloc((void**)&d_result, sizeof(float) * image_size));

// Copy data to the device
checkCudaErrors(hipMemcpy(d_image, hData, image_size, hipMemcpyHostToDevice));
checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_M), h_mask, mask_size));

// CUDA timing of event
hipEvent_t shared_start, shared_stop;
hipEventCreate(&shared_start);
hipEventCreate(&shared_stop);


// // Calculate grid dimensions
// int THREADS = 16;
// int BLOCKS = (N+THREADS-1)/THREADS;
// // printf("%d ", BLOCKS);
// // Dimension launch arguments
// dim3 block_dim(THREADS, THREADS);
// dim3 grid_dim(BLOCKS, BLOCKS);

// Calculate grid dimensions
// int THREADS = 16;
int BLOCKS = (width-1)/TILE_WIDTH+1;
// printf("%d ", BLOCKS);
// Dimension launch arguments
dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
dim3 dimGrid(BLOCKS, BLOCKS);

//-------------- Shared Convolution --------------//
// start the global memory kernel
    hipEventRecord(shared_start);
    shared_conv<<<dimGrid, dimBlock>>>(d_image, d_result, width, height);
    hipEventRecord(shared_stop); 
    hipEventSynchronize(shared_stop);
        
    float shared_elapsedTime = 0;
    hipEventElapsedTime(&shared_elapsedTime, shared_start, shared_stop);

    hipEventDestroy(shared_start);
    hipEventDestroy(shared_stop);

// Copy the h_result back to the CPU
checkCudaErrors(hipMemcpy(h_result, d_result, image_size, hipMemcpyDeviceToHost));
//   printf("Result image \n");
//   printArray(h_result, 10);

// Functional test
//  verify_result(hData, h_mask, h_result, N);
char outputFilename[1024];
strcpy(outputFilename, imagePath);
strcpy(outputFilename + strlen(imagePath) - 4, "_out.pgm");
sdkSavePGM(outputFilename, h_result, width, height);
printf("Wrote '%s'\n", outputFilename);

//-------------- CUDA Performance Metrics --------------//
printf("Shared Memory Time elpased: %3.6f ms \n", shared_elapsedTime);

// Free the memory we allocated
free(imagePath);
free(h_result);

//   checkCudaErrors(hipFree(h_mask));
//   checkCudaErrors(hipFree(d_mask));
checkCudaErrors(hipFree(d_image));
checkCudaErrors(hipFree(d_result));

return 0;
}
