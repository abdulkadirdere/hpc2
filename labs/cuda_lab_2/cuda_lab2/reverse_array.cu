// includes, system
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
 
// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);
 
// implement the kernel using global memory
__global__ void reverseArray(int *d_out, int *d_in){

}
// implement the kernel using shared memory
__global__ void reverseArray_shared(int *d_out, int *d_in){

}
 
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv){
    // pointer for host memory and size
    int *h_a;
    int dimA = 256 * 1024; // 256K elements (1MB total)
    // pointer for device memory
    int *d_b, *d_a;
    // define grid and block size
    int numThreadsPerBlock = 256;
    // Part 1: compute number of blocks needed based on array size and desired block size
    int numBlocks = dimA / numThreadsPerBlock;  
    // allocate host and device memory
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    h_a = (int *) malloc(memSize);
    hipMalloc( (void **) &d_a, memSize );
    hipMalloc( (void **) &d_b, memSize );
    // Initialize input array on host
    for (int i = 0; i < dimA; ++i){
        h_a[i] = i;
    }
    // Copy host array to device array
    hipMemcpy( d_a, h_a, memSize, hipMemcpyHostToDevice );
    // launch kernel
    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock);
    reverseArray<<< dimGrid, dimBlock >>>( d_b, d_a );
    // block until the device has completed
    hipDeviceSynchronize();
    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");
    // device to host copy
    hipMemcpy( h_a, d_b, memSize, hipMemcpyDeviceToHost );
    // Check for any CUDA errors
    checkCUDAError("memcpy");
    // verify the data returned to the host is correct
    for (int i = 0; i < dimA; i++){
        assert(h_a[i] == dimA - 1 - i );
    }
    // free device memory
    hipFree(d_a);
    hipFree(d_b);
    // free host memory
    free(h_a);
    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.
    printf("Global memory -- verified.\n");
    return 0;
}
void checkCUDAError(const char *msg){
    hipError_t err = hipGetLastError();
    if( hipSuccess != err){
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}
