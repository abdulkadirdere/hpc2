// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define N 256

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if( err != hipSuccess) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

///////////////////////////////////////////////////////////////////////////////
// Program main
///////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) {
    // pointers for host memory and size
    int *h_a = NULL, *h_b = NULL;
    // pointers for device memory
    int *d_a = NULL;

    // YOUR CODE GOES HERE
    // Allocate memory for N integers on the host for h_a and h_b
    // (a)
   


    // YOUR CODE GOES HERE
    // Allocate memory for N integers on the device for d_a
    // (b)
    


    // Initialize h_a to contain integers 0 .. N - 1
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
    }

    // Zero memory for h_b
    memset(h_b, 0, N * sizeof(int));

    // YOUR CODE GOES HERE
    // Transfer contents of h_a to to d_a
    // (c)
   

    // YOUR CODE GOES HERE
    // Transfer contents of d_a to h_b
    // (d)
  

    // Check for any CUDA errors
    checkCUDAError("memcpy");

    for (int i = 0; i < N; i++) {
        if (h_a[i] != h_b[i]) {
            printf("Test failed h_a[%d] != h_b[%d]\n", i, i);
            exit(1);
        }
    }

    // YOUR CODE GOES HERE
    // Free memory for host pointers h_a and h_b
    // (e)
    

    // YOUR CODE GOES HERE
    // Free memory for device pointer d_a
    // (f)
   
    printf("Test passed!\n");

    return 0;
}
